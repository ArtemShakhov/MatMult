#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "iostream"
#include <cstdlib>
#include <ctime>

#define BLOCK_SIZE  32          
#define N           640

using namespace std;



hipError_t addWithCuda(int *c, const int *a, const int *b, int time, int **ccpu);

__global__ void addKernel(const int *a, const int *b, int n, int * c)
{
	int   bx = blockIdx.x;     
	int   by = blockIdx.y;
	int   tx = threadIdx.x;        
	int   ty = threadIdx.y;
	float sum = 0;           
	int   ia = n * BLOCK_SIZE * by + n * ty;  
	int   ib = BLOCK_SIZE * bx + tx;

	
	for (int k = 0; k < n; k++)
		sum += a[ia + k] * b[ib + k * n];

	
	int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;

	c[ic + n * ty + tx] = sum;
}

int** CPU(int** a, int** b, int** c, int n) {
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
		{
			c[i][j] = 0;
			for (int k = 0; k < n; k++)
				c[i][j] += a[i][k] * b[k][j];
		}
	return c;
}

bool checkResult(int** a, int* b, int n) {
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
			if (a[i][j] != b[N * i + j])
				return false;
	return true;
}


int main( int argc, char *  argv[] )
{
	setlocale(LC_ALL, "Russian");
	
   

	// allocate host memory
	int** acpu;
	int** bcpu;
	int** ccpu;
	acpu = new int*[N];
	for (int i = 0; i < N; i++)
		acpu[i] = new int[N];

	bcpu = new int*[N];
	for (int i = 0; i < N; i++)
		bcpu[i] = new int[N];

	ccpu = new int*[N];
	for (int i = 0; i < N; i++)
		ccpu[i] = new int[N];

	int * a = new int[N*N];
	int * b = new int[N*N];
	int * c = new int[N*N];


	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			int	k = N * i + j;
			a[k]= (rand()) % 10 + 1;
			b[k] = (rand()) % 10 + 1;
			acpu[i][j] = a[k];
			bcpu[i][j] = b[k];
			ccpu[i][j] = 0;
		}

	}
	clock_t time;
	time = clock();
	ccpu = CPU(acpu, bcpu, ccpu, N);
	time = clock() - time;
	


    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, time, ccpu);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    
	
	

    
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, int time, int **ccpu)
{
    int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	int numBytes = N * N * sizeof(int);
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        cout << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";
        goto Error;
    }
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(N / threads.x, N / threads.y);

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, numBytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, numBytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, numBytes);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	hipEvent_t start, stop;
	float gpuTime = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// asynchronously issue work to the GPU (all to stream 0)
	hipEventRecord(start, 0);

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, numBytes, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, numBytes, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<blocks, threads>>>(dev_a, dev_b, N, dev_c);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, numBytes, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);
	
	
	if (checkResult(ccpu, c, N))
	{
		cout << "Âðåìÿ ðàáîòû íà CPU: " << time * 1000.0 / CLOCKS_PER_SEC << " ìèëëèñåêóíä" << endl;
		cout << "Âðåìÿ ðàáîòû íà GPU: " << gpuTime << " ìèëëèñåêóíä";
	}

Error:
	hipEventDestroy(start);
	hipEventDestroy(stop);
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

	delete a;
	delete b;
	delete c;

    
    return cudaStatus;
}
